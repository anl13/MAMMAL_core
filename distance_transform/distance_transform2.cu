#include "hip/hip_runtime.h"

#include "distance_transform_cuda.h"
#include <hip/hip_runtime.h>
#include <>


#define BLOCK_SIZE 256

__global__ void euclidian_distance_transform_kernel(
	const unsigned char* img, float* dist, int w, int h)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int N = w * h;

	if (i >= N)
	{
		return;
	}

	int cx = i % w;
	int cy = i / w;

	float minv = INFINITY;

	if (img[i] > 0)
	{
		minv = 0.0f;
	}
	else
	{
		for (int j = 0; j < N; j++)
		{
			if (img[j] > 0)
			{
				int x = j % w;
				int y = j / w;
				float d = sqrtf(powf(float(x - cx), 2.0f) + powf(float(y - cy), 2.0f));
				if (d < minv) minv = d;
			}
		}
	}

	dist[i] = minv;
}

void euclidian_distance_transform(unsigned char* img, float* dist, int w, int h) {

	hipError_t err;
	unsigned char *d_img;
	hipMalloc((void**)&d_img, w*h * sizeof(unsigned char));
	hipMemcpy(d_img, img, w*h * sizeof(unsigned char), hipMemcpyHostToDevice);

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA ERROR: %s\n", hipGetErrorString(err));
	}

	float* d_dist;
	hipMalloc((void**)&d_dist, w*h * sizeof(float));
	//hipMemset(d_dist, 0, w*h*sizeof(float));

	dim3 block(BLOCK_SIZE, 1, 1);

	int gx = (w*h + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 grid(gx, 1);

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA ERROR: %s\n", hipGetErrorString(err));
	}

	euclidian_distance_transform_kernel << <grid, block >> > (d_img, d_dist, w, h);
	hipDeviceSynchronize();

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA ERROR: %s\n", hipGetErrorString(err));
	}

	hipMemcpy(dist, d_dist, w*h * sizeof(float), hipMemcpyDeviceToHost);

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA ERROR: %s\n", hipGetErrorString(err));
	}

	hipFree(d_img);
	hipFree(d_dist);
}