#include "gpuutils.h"

#include <stdlib.h>
#include <time.h> 
#include <stdio.h> 

void cuda_set_device(int n)
{
	hipError_t status = hipSetDevice(n); 
	check_error(status); 
}

int cuda_get_device()
{
	int n = 0; 
	hipError_t status = hipGetDevice(&n); 
	check_error(status); 
	return n;
}

void check_error(hipError_t status)
{
	if (status != hipSuccess)
	{
		const char *s = hipGetErrorString(status); 
		char buffer[256]; 
		printf("CUDA error: %s\n", s); 
		snprintf(buffer, 256, "CUDA error: %s", s); 
	}
}